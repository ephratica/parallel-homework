#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>
#define PI acos(-1)

struct timeval tv_begin, tv_end;

void swap(hipDoubleComplex &a,hipDoubleComplex &b){
    hipDoubleComplex temp=a;
    a=b;
    b=temp;
    return;
}

void change(hipDoubleComplex *y, int len) {
    int *rev=new int[len];
    rev[0]=0;
    for (int i = 0; i < len; ++i) {
        rev[i] = rev[i >> 1] >> 1;
        if (i & 1) {
            rev[i] |= len >> 1;
        }
    }
    for (int i = 0; i < len; ++i) {
        if (i < rev[i]) {
            swap(y[i], y[rev[i]]);
        }
    }
    return;
}

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void initWith(hipDoubleComplex* pow,int h,int on){
    pow[0]= make_hipDoubleComplex(1,0);
    hipDoubleComplex wn= make_hipDoubleComplex(cos(2 * PI / h),sin(on * 2 * PI / h));
    for(int i=1;i<h/2;i++){
        pow[i]= hipCmul(pow[i-1],wn);
    }
}


__global__
void fft_cuda(hipDoubleComplex* y,hipDoubleComplex* temp,int n,int h,hipDoubleComplex* pow)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;//开始位置的索引
    int stride = blockDim.x * gridDim.x;//跳过其他由线程处理的索引


    for(int i = index; i < n; i += stride){
        int j=(i/h)*h;
        if(i-j<h/2) {
            hipDoubleComplex w= pow[i-j];
            //if(h==4)printf("fft_cuda_before\nw:%lf %lf,y[%d]:%lf %lf,y[%d]:%lf %lf\n",w.x,w.y,i,y[i].x,y[i].y,i+h/2,y[i+h/2].x,y[i+h/2].y);
            hipDoubleComplex u=temp[i];
            hipDoubleComplex t= hipCmul(temp[i+h/2],w);
            y[i]= hipCadd(u,t);
            y[i+h/2]= hipCsub(u,t);
            //if(h==4)printf("fft_cuda_after\nw:%lf %lf,y[%d]:%lf %lf,y[%d]:%lf %lf\n",w.x,w.y,i,y[i].x,y[i].y,i+h/2,y[i+h/2].x,y[i+h/2].y);
        }
    }
}

void check(hipDoubleComplex* y,hipDoubleComplex* yy,int n){
    for (int i = 0; i < n; i++) {
        if(abs(y[i].x-yy[i].x)>(0.00001)||abs(y[i].y-yy[i].y)>(0.00001)){
            printf("wrong\n");
            printf("%d\n",i);
            printf("%lf %lf\n",y[i].x,y[i].y);
            printf("%lf %lf\n",yy[i].x,yy[i].y);
            printf("%lf,%lf",abs(y[i].x-yy[i].x),abs(y[i].y-yy[i].y));
            return;
        }
    }
    printf("correct\n");
}

void Print(hipDoubleComplex* y,hipDoubleComplex* yy,int n){
    for (int i = 0; i < n; i++) {
        printf("%f %f\n",y[i].x,y[i].y);
        printf("%f %f\n",yy[i].x,yy[i].y);
        printf("&&&&&&&&&&&&&&&&\n");
    }
}

void fft(hipDoubleComplex *y, int len, int on) {
    change(y, len);
    for (int h = 2; h <= len; h <<= 1) {
        hipDoubleComplex wn=make_hipDoubleComplex(cos(2 * PI / h), sin(on * 2 * PI / h));
        for (int j = 0; j < len; j += h) {
            hipDoubleComplex w=make_hipDoubleComplex(1, 0);
            for (int k = j; k < j + h / 2; k++) {
                hipDoubleComplex u = y[k];
                hipDoubleComplex t = hipCmul(w,y[k+h/2]);
                //if(h==4)printf("fft_before\nw:%lf %lf,y[%d]:%lf %lf,y[%d]:%lf %lf\n",w.x,w.y,k,y[k].x,y[k].y,k+h/2,y[k+h/2].x,y[k+h/2].y);
                y[k] = hipCadd(u,t);
                y[k + h / 2] = hipCsub(u,t);
                w = hipCmul(w,wn);
                //if(h==4)printf("fft_after\nw:%lf %lf,y[%d]:%lf %lf,y[%d]:%lf %lf\n",w.x,w.y,k,y[k].x,y[k].y,k+h/2,y[k+h/2].x,y[k+h/2].y);
            }
        }
    }
    if (on == -1) {
        for (int i = 0; i < len; i++) {
            y[i].x /= len;y[i].y/=len;
        }
    }
}

int main()
{
    const int N = 1<<25;
    size_t size = N * sizeof(hipDoubleComplex);

    hipDoubleComplex* y;
    hipDoubleComplex* yy;
    hipDoubleComplex* temp;
    hipDoubleComplex* pow;

    //cudaMallocManaged(&a, size)的作用相当于a = (float *)malloc(size);
    //区别在于申请的是gpu可以访问的内存
    checkCuda( hipMallocManaged(&y, size) );
    checkCuda( hipMallocManaged(&yy, size) );
    checkCuda( hipMallocManaged(&temp, size) );
    checkCuda( hipMallocManaged(&pow, size) );

    //选定网络上的block大小
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    for (int i = 0; i < N; i++) {
        y[i].x = yy[i].x = rand();
        y[i].y = yy[i].y = rand();
    }

    gettimeofday(&tv_begin,NULL);
    fft(yy,N,1);
    gettimeofday(&tv_end,NULL);
    long long sb=tv_begin.tv_sec*(1e6)+tv_begin.tv_usec,se=tv_end.tv_sec*(1e6)+tv_end.tv_usec;
    printf("normal: %lld\n",se-sb);

    gettimeofday(&tv_begin,NULL);
    change(y,N);
    for (int h=2;h<=N;h<<=1){
        initWith(pow,h,1);
        memcpy(temp,y, size);
        //类似串行程序的函数调用，我们把这种“函数调用”称为“a kernel is launched”
        //<<<numberOfBlocks, threadsPerBlock>>>顾名思义，就是块数和线程数
        fft_cuda<<<numberOfBlocks, threadsPerBlock>>>(y, temp,N, h, pow);

        //检查错误
        //checkCuda( cudaGetLastError() );
        //cpu程序不会继续执行直到gpu的kernel全部完成
        //checkCuda( cudaDeviceSynchronize() );
        hipDeviceSynchronize();
    }
    gettimeofday(&tv_end,NULL);
    sb=tv_begin.tv_sec*(1e6)+tv_begin.tv_usec,se=tv_end.tv_sec*(1e6)+tv_end.tv_usec;
    printf("cuda: %lld\n",se-sb);



    //Print(y,yy,8);
    check(y,yy,N);

    //释放内存
    checkCuda( hipFree(y) );
    checkCuda( hipFree(yy) );
    checkCuda( hipFree(temp) );
    checkCuda( hipFree(pow) );
}
